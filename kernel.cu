#include "hip/hip_runtime.h"
﻿#define CUDA_LIBRARY_KERNEL

#include "cudalibrary.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#ifndef CUDAMEMORY_H
#define CUDAMEMORY_H

long cudaMemoryResetDevice()
{
	hipError_t cudaStatus;

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemorySetDevice(unsigned long deviceID)
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(deviceID);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryAllocateMemory(void** result, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)result, length);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryAllocateMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemorySetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(devicememory, hostmemory, length, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemorySetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryGetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(hostmemory, devicememory, length, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryGetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryCopyDeviceMemory(void* devicedestination, void* devicesource, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(devicedestination, devicesource, length, hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryCopyDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryDeviceSynchronize()
{
	hipError_t cudaStatus;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryDeviceSynchronize failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryFreeMemory(void* source)
{
	hipError_t cudaStatus;

	cudaStatus = hipFree(source);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryFreeMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

#endif // CUDAMEMORY_H

#ifndef CUDAINDEXMAP_H
#define CUDAINDEXMAP_H

#include "cudalibrary.h"

long cudaIndexMapCreateTypeCopy(cudaIndexMap* parent, typeIndexMap* source)
{
	unsigned long* data;

	if ((parent == 0) || (source == 0))
		return 0;

	if (cudaMemoryAllocateMemory((void**)&data, source->indexSize))
	{
		cudaMemorySetDeviceMemory(data, source->indexData, source->indexSize);

		parent->indexWidth = source->indexWidth;
		parent->indexHeight = source->indexHeight;
		parent->indexLength = source->indexLength;
		parent->indexSize = source->indexSize;
		parent->indexData = data;

		return 1;
	}

	parent->indexWidth = 0;
	parent->indexHeight = 0;
	parent->indexLength = 0;
	parent->indexSize = 0;
	parent->indexData = 0;

	return 0;
}

long cudaIndexMapDestroy(cudaIndexMap* parent)
{
	cudaMemoryFreeMemory(parent->indexData);

	parent->indexWidth = 0;
	parent->indexHeight = 0;
	parent->indexLength = 0;
	parent->indexSize = 0;
	parent->indexData = 0;

	return 1;
}

#endif //  CUDAINDEXMAP_H

#ifndef CUDANEURALARRAY_H
#define CUDANEURALARRAY_H

#include "cudalibrary.h"

#include <stdlib.h>

long cudaNeuralArrayCreate(cudaNeuralArray* parent, unsigned long width, unsigned long height, unsigned long depth)
{
	cudaNeuralUnit* data;
	unsigned long   length;
	unsigned long   datalength;

	if (parent == 0)
		return 0;

	typeShapeAssign(&parent->arrayShape, width, height, depth);
	length = typeShapeGetLength(&parent->arrayShape);
	datalength = sizeof(cudaNeuralUnit) * length;

	if (cudaMemoryAllocateMemory((void**)&data, datalength))
	{
		parent->arrayLength = length;
		parent->arraySize = datalength;
		parent->arrayData = data;

		return 1;
	}

	typeShapeZero(&parent->arrayShape);
	parent->arrayLength = 0;
	parent->arraySize = 0;
	parent->arrayData = 0;

	return 0;
}

long cudaNeuralArraySetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
	if ((parent == 0) || (buffer == 0))
		return 0;

	if (buffersize != parent->arrayLength)
		return 0;

	return cudaMemorySetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayGetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
	if ((parent == 0) || (buffer == 0))
		return 0;

	if (buffersize != parent->arrayLength)
		return 0;

	return cudaMemoryGetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayCopy(cudaNeuralArray* parent, cudaNeuralArray* source)
{
	if ((parent == 0) || (source == 0))
		return 0;

	if (parent->arrayLength != source->arrayLength)
		return 0;

	return cudaMemoryCopyDeviceMemory(parent->arrayData, source->arrayData, parent->arrayLength * sizeof(cudaNeuralUnit));
}

long cudaNeuralArraySetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
	unsigned long   I, J;
	unsigned long   A, B, C;
	unsigned long   length;
	unsigned char* color;
	cudaNeuralUnit* buffer;

	if ((parent == 0) || (imagedata == 0))
		return 0;

	if (typeShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
		return 0;

	color = (unsigned char*)imagedata;
	buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
	if (buffer == 0)
		return 0;

	length = imagewidth * imageheight;
	A = length * 0;
	B = length * 1;
	C = length * 2;

	J = 0;
	for (I = 0; I < length; I++)
	{
		buffer[A + I] = (cudaNeuralUnit)color[J + 0] / 255;
		buffer[B + I] = (cudaNeuralUnit)color[J + 1] / 255;
		buffer[C + I] = (cudaNeuralUnit)color[J + 2] / 255;

		J += 4;
	}

	cudaNeuralArraySetData(parent, buffer, parent->arrayLength);

	free(buffer);

	return 1;
}

double neural_maxout(double a)
{
	if (a < 0)
		return 0;

	if (a > 1)
		return 1;

	return a;
}

long cudaNeuralArrayGetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
	unsigned long   I, J;
	unsigned long   A, B, C;
	unsigned long   length;
	unsigned char* color;
	cudaNeuralUnit* buffer;

	if ((parent == 0) || (imagedata == 0))
		return 0;

	if (typeShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
		return 0;

	color = (unsigned char*)imagedata;
	buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
	if (buffer == 0)
		return 0;

	cudaNeuralArrayGetData(parent, buffer, parent->arrayLength);

	length = imagewidth * imageheight;
	A = length * 0;
	B = length * 1;
	C = length * 2;

	J = 0;
	for (I = 0; I < length; I++)
	{
		color[J + 0] = (unsigned char)(neural_maxout(buffer[A + I]) * 255);
		color[J + 1] = (unsigned char)(neural_maxout(buffer[B + I]) * 255);
		color[J + 2] = (unsigned char)(neural_maxout(buffer[C + I]) * 255);

		J += 4;
	}

	free(buffer);

	return 1;
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillZero(cudaNeuralUnit* result, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = 0;
}

long cudaNeuralArrayFillZero(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelFillZero << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillOnes(cudaNeuralUnit* result, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = 1;
}

long cudaNeuralArrayFillOnes(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelFillOnes << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void kernelFillRandom(cudaNeuralUnit* result, unsigned long seed, unsigned long length)
{
	hiprandState    state;
	unsigned long  I;

	hiprand_init(seed, 0, 0, &state);

	for (I = 0; I < length; I++)
	{
		result[I] = hiprand_uniform(&state);
	}
}

long cudaNeuralArrayFillRandom(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	kernelFillRandom << < 1, 1 >> > (result->arrayData, clock(), result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelAdd(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] + value2[I];
}

long cudaNeuralArrayAdd(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelAdd << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelSubtract(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] - value2[I];
}

long cudaNeuralArraySubtract(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelSubtract << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelMultiply(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] * value2[I];
}

long cudaNeuralArrayMultiply(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelMultiply << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelDivide(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	if (value2[I] != 0)
		result[I] = value1[I] * value2[I];
	else
		result[I] = 0;
}

long cudaNeuralArrayDivide(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelDivide << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__managed__ cudaNeuralUnit     cudaRegister1;

__global__ void kernelGetMeanSquaredError(cudaNeuralUnit* source, cudaNeuralUnit* target, unsigned long length)
{
	unsigned long   I;
	cudaNeuralUnit  sum;

	sum = 0;
	for (I = 0; I < length; I++)
	{
		sum += (source[I] - target[I]) * (source[I] - target[I]);
	}

	if (length > 1)
		sum = sum / length;

	cudaRegister1 = sum;
}

long cudaNeuralArrayGetMeanSquaredError(cudaNeuralArray* source, cudaNeuralArray* target, cudaNeuralUnit* result)
{
	if ((source == 0) || (target == 0) || (result == 0))
		return 0;

	if (source->arrayLength != target->arrayLength)
		return 0;

	kernelGetMeanSquaredError << < 1, 1 >> > (source->arrayData, target->arrayData, source->arrayLength);

	cudaMemoryDeviceSynchronize();

	*result = cudaRegister1;

	return 1;
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMapCopy2D(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  counter;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	counter = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index];
			counter++;
		}

		K += indexwidth;
	}

	if (counter > 1)
	{
		sum /= counter;
	}

	result[resultindex + I] = sum;
}

long cudaNeuralArrayIndexMapCopy2D(cudaNeuralArray* result, cudaNeuralArray* source, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMapCopy2D << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);


	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMapPooling2DForward(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (fabsf(source[sourceindex + index]) > fabsf(sum))
				sum = source[sourceindex + index];
		}

		K += indexwidth;
	}


	result[resultindex + I] = sum;
}

long cudaNeuralArrayIndexMapPooling2DForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMapPooling2DForward << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMapPooling2DReverse(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* outputs,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  counter;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	counter = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (fabsf(outputs[sourceindex + index]) > fabsf(sum))
			{
				sum = outputs[sourceindex + index];
				counter = J;
			}
		}

		K += indexwidth;
	}

	K = I;
	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (counter == J)
				source[sourceindex + index] = result[resultindex + I];
			else
				source[sourceindex + index] = 0;
		}

		K += indexwidth;
	}
}

long cudaNeuralArrayIndexMapPooling2DReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* outputs, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (outputs == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	if (typeShapeCompare(&source->arrayShape, &outputs->arrayShape))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMapPooling2DReverse << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		outputs->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__device__ cudaNeuralUnit indexMapConvolution2DValue(
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long  indexoffset,
	unsigned long  sourceindex,
	unsigned long  sourcelength,
	unsigned long  filterindex,
	unsigned long  filterlength)
{
	unsigned long  I, J;
	unsigned long  index;
	cudaNeuralUnit sum;

	sum = 0;
	J = indexoffset;
	for (I = 0; I < indexheight; I++)
	{
		index = indexmap[J];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index] * filter[filterindex + I];
		}

		J += indexwidth;
	}

	return sum;
}

__global__ void kernelArrayIndexMapConvolution2DForward(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I;
	unsigned long  J = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  K = blockDim.y * blockIdx.y + threadIdx.y;
	cudaNeuralUnit sum;

	if (J >= resultcount)
		return;
	if (K >= resultlength)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = 0;
	unsigned long   filterindex = (J * sourcecount) * filterlength;

	sum = 0;
	for (I = 0; I < sourcecount; I++)
	{
		sum += indexMapConvolution2DValue(
			source,
			filter,
			indexwidth,
			indexheight,
			indexmap,
			K,
			sourceindex,
			sourcelength,
			filterindex,
			filterlength);

		sourceindex += sourcelength;
		filterindex += filterlength;
	}

	result[resultindex + K] = sum;
}

long cudaNeuralArrayConvolution2DForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != resultlength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultcount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultlength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMapConvolution2DForward << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMapConvolution2DReverse(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J;
	unsigned long  K = blockDim.y * blockIdx.y + threadIdx.y;
	cudaNeuralUnit sum;

	if (I >= sourcecount)
		return;
	if (K >= sourcelength)
		return;

	unsigned long   resultindex = 0;
	unsigned long   sourceindex = I * sourcelength;
	unsigned long   filterindex = I * filterlength;
	unsigned long   filterjump = sourcecount * filterlength;

	sum = 0;
	for (J = 0; J < resultcount; J++)
	{
		sum += indexMapConvolution2DValue(
			result,
			filter,
			indexwidth,
			indexheight,
			indexmap,
			K,
			resultindex,
			resultlength,
			filterindex,
			filterlength);

		resultindex += resultlength;
		filterindex += filterjump;
	}

	source[sourceindex + K] = sum;
}

long cudaNeuralArrayConvolution2DReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != sourcelength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(sourcecount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(sourcelength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMapConvolution2DReverse << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMapConvolution2DGetDerivatives(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K = blockDim.z * blockIdx.z + threadIdx.z;

	if (I >= sourcecount)
		return;
	if (J >= resultcount)
		return;
	if (K >= filterlength)
		return;

	unsigned long   sourceindex = I * sourcelength;
	unsigned long   resultindex = J * resultlength;
	unsigned long   filterindex = (J * sourcecount + I) * filterlength;

	unsigned long  X, Y;
	unsigned long  index;
	unsigned long  counter;
	cudaNeuralUnit sum;

	sum = 0;
	counter = 0;
	Y = K * indexwidth;
	for (X = 0; X < indexwidth; X++)
	{
		index = indexmap[Y + X];
		if ((index != TYPE_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index] * result[resultindex + X];
			counter++;
		}
	}

	if (counter > 1)
		sum = sum / counter;

	filter[filterindex + K] = sum;

}

long cudaNeuralArrayConvolution2DGetDerivatives(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != resultlength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(8, 8, 8);
	dim3 blocksPerGrid(
		(sourcecount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y,
		(filterlength + threadsPerBlock.z - 1) / threadsPerBlock.z);

	kernelArrayIndexMapConvolution2DGetDerivatives << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixForward(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I;
	unsigned long  J = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  K;
	cudaNeuralUnit sum;

	if (J >= resultlength)
		return;

	sum = 0;
	K = J * sourcelength;
	for (I = 0; I < sourcelength; I++)
	{
		sum += source[I] * weights[K + I];
	}

	result[J] = sum;
}

long cudaNeuralArrayMatrixForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayMatrixForward << < blocksPerGrid, threadsPerBlock >> > 
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixReverse(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J;
	unsigned long  K;
	cudaNeuralUnit sum;

	if (I >= sourcelength)
		return;

	sum = 0;
	K = I;
	for (J = 0; J < resultlength; J++)
	{
		sum += result[J] * weights[K];

		K += sourcelength;
	}

	source[I] = sum;
}

long cudaNeuralArrayMatrixReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((source->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayMatrixReverse << < blocksPerGrid, threadsPerBlock >> >
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}


//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixGetDerivatives(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;

	if (I >= sourcelength)
		return;
	if (J >= resultlength)
		return;

	weights[J * sourcelength + I] = result[J] * source[I];
}

long cudaNeuralArrayMatrixGetDerivatives(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(source->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(result->arrayLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayMatrixGetDerivatives << < blocksPerGrid, threadsPerBlock >> >
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelReverse(cudaNeuralUnit* result, cudaNeuralUnit* source, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = source[length - I - 1];
}

long cudaNeuralArrayReverse(cudaNeuralArray* result, cudaNeuralArray* source)
{
	if ((result == 0) || (source == 0))
		return 0;

	if (result->arrayLength != source->arrayLength)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelReverse << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, source->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

long cudaNeuralArrayDestroy(cudaNeuralArray* parent)
{
	if (parent == 0)
		return 0;

	cudaMemoryFreeMemory((void*)parent->arrayData);

	parent->arrayLength = 0;
	parent->arraySize = 0;
	parent->arrayData = 0;

	return 1;
}

#endif // CUDANEURALARRAY_H

