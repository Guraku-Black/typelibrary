#include "hip/hip_runtime.h"
﻿#define CUDA_LIBRARY_KERNEL

#include "cudalibrary.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#ifndef CUDAMEMORY_H
#define CUDAMEMORY_H

long cudaMemoryResetDevice()
{
	hipError_t cudaStatus;

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemorySetDevice(unsigned long deviceID)
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(deviceID);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryAllocateMemory(void** result, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)result, length);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryAllocateMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemorySetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(devicememory, hostmemory, length, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemorySetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryGetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(hostmemory, devicememory, length, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryGetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryCopyDeviceMemory(void* devicedestination, void* devicesource, unsigned long length)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(devicedestination, devicesource, length, hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryCopyDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryDeviceSynchronize()
{
	hipError_t cudaStatus;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryDeviceSynchronize failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

long cudaMemoryFreeMemory(void* source)
{
	hipError_t cudaStatus;

	cudaStatus = hipFree(source);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaMemoryFreeMemory failed! %s\n", hipGetErrorString(cudaStatus));

		return 0;
	}

	return 1;
}

#endif // CUDAMEMORY_H

#ifndef CUDASHAPE_H
#define CUDASHAPE_H

#include "cudalibrary.h"

void cudaShapeZero(cudaShape* parent)
{
	parent->shapeWidth = 0;
	parent->shapeHeight = 0;
	parent->shapeDepth = 0;
}

void cudaShapeAssign(cudaShape* parent, unsigned long X, unsigned long Y, unsigned long Z)
{
	parent->shapeWidth = X;
	parent->shapeHeight = Y;
	parent->shapeDepth = Z;
}

long cudaShapeCompare(cudaShape* parent, cudaShape* target)
{
	if ((parent->shapeWidth != target->shapeWidth) ||
		(parent->shapeHeight != target->shapeHeight) ||
		(parent->shapeDepth != target->shapeDepth))
		return 0;

	return 1;
}

long cudaShapeCompareValue(cudaShape* parent, unsigned long X, unsigned long Y, unsigned long Z)
{
	if ((parent->shapeWidth != X) ||
		(parent->shapeHeight != Y) ||
		(parent->shapeDepth != Z))
		return 0;

	return 1;
}

unsigned long cudaShapeGetLength(cudaShape* parent)
{
	return (parent->shapeWidth * parent->shapeHeight * parent->shapeDepth);
}

#endif // CUDASHAPE_H

#ifndef CUDAINDEXMAP_H
#define CUDAINDEXMAP_H

#include "cudalibrary.h"

long cudaIndexMapCreateCopy(cudaIndexMap* parent, unsigned long width, unsigned long height, unsigned long* source)
{
	unsigned long  length;
	unsigned long  datalength;
	unsigned long* data;

	if ((parent == 0) || (source == 0))
		return 0;

	length = width * height;
	datalength = length * sizeof(unsigned long);
	if (cudaMemoryAllocateMemory((void**)&data, datalength))
	{
		cudaMemorySetDeviceMemory(data, source, datalength);

		parent->indexWidth = width;
		parent->indexHeight = height;
		parent->indexLength = length;
		parent->indexSize = datalength;
		parent->indexData = data;

		return 1;
	}

	parent->indexWidth = 0;
	parent->indexHeight = 0;
	parent->indexLength = 0;
	parent->indexSize = 0;
	parent->indexData = 0;

	return 0;
}

long cudaIndexMapDestroy(cudaIndexMap* parent)
{
	cudaMemoryFreeMemory(parent->indexData);

	parent->indexWidth = 0;
	parent->indexHeight = 0;
	parent->indexLength = 0;
	parent->indexSize = 0;
	parent->indexData = 0;

	return 1;
}

#endif //  CUDAINDEXMAP_H

#ifndef CUDANEURALARRAY_H
#define CUDANEURALARRAY_H

#include "cudalibrary.h"

#include <stdlib.h>

long cudaNeuralArrayCreate(cudaNeuralArray* parent, unsigned long width, unsigned long height, unsigned long depth)
{
	cudaNeuralUnit* data;
	unsigned long   length;
	unsigned long   datalength;

	if (parent == 0)
		return 0;

	cudaShapeAssign(&parent->arrayShape, width, height, depth);
	length = cudaShapeGetLength(&parent->arrayShape);
	datalength = sizeof(cudaNeuralUnit) * length;

	if (cudaMemoryAllocateMemory((void**)&data, datalength))
	{
		parent->arrayLength = length;
		parent->arraySize = datalength;
		parent->arrayData = data;

		return 1;
	}

	cudaShapeZero(&parent->arrayShape);
	parent->arrayLength = 0;
	parent->arraySize = 0;
	parent->arrayData = 0;

	return 0;
}

long cudaNeuralArraySetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
	if ((parent == 0) || (buffer == 0))
		return 0;

	if (buffersize != parent->arrayLength)
		return 0;

	return cudaMemorySetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayGetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
	if ((parent == 0) || (buffer == 0))
		return 0;

	if (buffersize != parent->arrayLength)
		return 0;

	return cudaMemoryGetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayCopy(cudaNeuralArray* parent, cudaNeuralArray* source)
{
	if ((parent == 0) || (source == 0))
		return 0;

	if (parent->arrayLength != source->arrayLength)
		return 0;

	return cudaMemoryCopyDeviceMemory(parent->arrayData, source->arrayData, parent->arrayLength * sizeof(cudaNeuralUnit));
}

long cudaNeuralArraySetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
	unsigned long   I, J;
	unsigned long   A, B, C;
	unsigned long   length;
	unsigned char* color;
	cudaNeuralUnit* buffer;

	if ((parent == 0) || (imagedata == 0))
		return 0;

	if (cudaShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
		return 0;

	color = (unsigned char*)imagedata;
	buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
	if (buffer == 0)
		return 0;

	length = imagewidth * imageheight;
	A = length * 0;
	B = length * 1;
	C = length * 2;

	J = 0;
	for (I = 0; I < length; I++)
	{
		buffer[A + I] = (cudaNeuralUnit)color[J + 0] / 255;
		buffer[B + I] = (cudaNeuralUnit)color[J + 1] / 255;
		buffer[C + I] = (cudaNeuralUnit)color[J + 2] / 255;

		J += 4;
	}

	cudaNeuralArraySetData(parent, buffer, parent->arrayLength);

	free(buffer);

	return 1;
}

double neural_maxout(double a)
{
	if (a < 0)
		return 0;

	if (a > 1)
		return 1;

	return a;
}

long cudaNeuralArrayGetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
	unsigned long   I, J;
	unsigned long   A, B, C;
	unsigned long   length;
	unsigned char* color;
	cudaNeuralUnit* buffer;

	if ((parent == 0) || (imagedata == 0))
		return 0;

	if (cudaShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
		return 0;

	color = (unsigned char*)imagedata;
	buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
	if (buffer == 0)
		return 0;

	cudaNeuralArrayGetData(parent, buffer, parent->arrayLength);

	length = imagewidth * imageheight;
	A = length * 0;
	B = length * 1;
	C = length * 2;

	J = 0;
	for (I = 0; I < length; I++)
	{
		color[J + 0] = (unsigned char)(neural_maxout(buffer[A + I]) * 255);
		color[J + 1] = (unsigned char)(neural_maxout(buffer[B + I]) * 255);
		color[J + 2] = (unsigned char)(neural_maxout(buffer[C + I]) * 255);

		J += 4;
	}

	free(buffer);

	return 1;
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillZero(cudaNeuralUnit* result, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = 0;
}

long cudaNeuralArrayFillZero(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelFillZero << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillOnes(cudaNeuralUnit* result, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = 1;
}

long cudaNeuralArrayFillOnes(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelFillOnes << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void kernelFillRandomUniform(cudaNeuralUnit* result, unsigned long seed, unsigned long length)
{
	hiprandState    state;
	unsigned long  I;

	hiprand_init(seed, 0, 0, &state);

	for (I = 0; I < length; I++)
	{
		result[I] = hiprand_uniform(&state);
	}
}

long cudaNeuralArrayFillRandomUniform(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	kernelFillRandomUniform << < 1, 1 >> > (result->arrayData, clock(), result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillRandomNormal(cudaNeuralUnit* result, unsigned long seed, unsigned long length)
{
	hiprandState    state;
	unsigned long  I;

	hiprand_init(seed, 0, 0, &state);

	for (I = 0; I < length; I++)
	{
		result[I] = hiprand_normal(&state);
	}
}

long cudaNeuralArrayFillRandomNormal(cudaNeuralArray* result)
{
	if (result == 0)
		return 0;

	kernelFillRandomNormal << < 1, 1 >> > (result->arrayData, clock(), result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillRandomXavier(cudaNeuralUnit* result, unsigned long seed, unsigned long fanIn, unsigned long fanOut, unsigned long length)
{
	hiprandState    state;
	cudaNeuralUnit value;
	cudaNeuralUnit avg;
	unsigned long  I;

	hiprand_init(seed, 0, 0, &state);

	for (I = 0; I < length; I++)
	{
	    value = hiprand_normal(&state);

		avg = ((double)fanIn + fanOut) / 2;
		result[I] = value * (1.0 / sqrt(avg)); //  XAVIER
		//result[I] = value * (2.0 / sqrt((double)fanIn)); //  HE
		//result[I] = value * (1.0 / sqrt((double)fanIn)); //  LeCun 

	}
}

long cudaNeuralArrayFillRandomXavier(cudaNeuralArray* result, unsigned long fanIn, unsigned long fanOut)
{
	if (result == 0)
		return 0;

	kernelFillRandomXavier << < 1, 1 >> > (result->arrayData, clock(), fanIn, fanOut, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillOneHot(cudaNeuralUnit* result, unsigned long index, unsigned long length)
{
	unsigned long  I;

	for (I = 0; I < length; I++)
	{
		if (I == index)
			result[I] = 1;
		else
			result[I] = 0;
	}
}

long cudaNeuralArrayFillOneHot(cudaNeuralArray* result, unsigned long index)
{
	if (result == 0)
		return 0;

	kernelFillOneHot << < 1, 1 >> > (result->arrayData, index, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelAdd(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] + value2[I];
}

long cudaNeuralArrayAdd(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelAdd << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelSubtract(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] - value2[I];
}

long cudaNeuralArraySubtract(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelSubtract << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelMultiply(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	result[I] = value1[I] * value2[I];
}

long cudaNeuralArrayMultiply(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelMultiply << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelDivide(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	if (value2[I] != 0)
		result[I] = value1[I] * value2[I];
	else
		result[I] = 0;
}

long cudaNeuralArrayDivide(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
	if ((result == 0) || (value1 == 0) || (value2 == 0))
		return 0;

	if ((result->arrayLength != value1->arrayLength) ||
		(result->arrayLength != value2->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelDivide << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__managed__ cudaNeuralUnit     cudaRegister1;

__global__ void kernelArrayGetMeanSquaredError(cudaNeuralUnit* source, cudaNeuralUnit* target, unsigned long length)
{
	unsigned long   I;
	cudaNeuralUnit  sum;

	sum = 0;
	for (I = 0; I < length; I++)
	{
		sum += (source[I] - target[I]) * (source[I] - target[I]);
	}

	if (length > 1)
		sum = sum / length;

	cudaRegister1 = sum;
}

long cudaNeuralArrayGetMeanSquaredError(cudaNeuralArray* source, cudaNeuralArray* target, cudaNeuralUnit* result)
{
	if ((source == 0) || (target == 0) || (result == 0))
		return 0;

	if (source->arrayLength != target->arrayLength)
		return 0;

	kernelArrayGetMeanSquaredError << < 1, 1 >> > (source->arrayData, target->arrayData, source->arrayLength);

	cudaMemoryDeviceSynchronize();

	*result = cudaRegister1;

	return 1;
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMap2DCopy(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  counter;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	counter = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index];
			counter++;
		}

		K += indexwidth;
	}

	if (counter > 1)
	{
		sum /= counter;
	}

	result[resultindex + I] = sum;
}

long cudaNeuralArrayIndexMap2DCopy(cudaNeuralArray* result, cudaNeuralArray* source, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMap2DCopy << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);


	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMap2DPoolingForward(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (fabsf(source[sourceindex + index]) > fabsf(sum))
				sum = source[sourceindex + index];
		}

		K += indexwidth;
	}


	result[resultindex + I] = sum;
}

long cudaNeuralArrayIndexMap2DPoolingForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMap2DPoolingForward << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMap2DPoolingReverse(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* outputs,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount)
{
	cudaNeuralUnit sum;
	unsigned long  counter;
	unsigned long  index;

	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K;

	if (I >= resultlength)
		return;

	if (J >= resultcount)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = J * sourcelength;

	sum = 0;
	counter = 0;
	K = I;

	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (fabsf(outputs[sourceindex + index]) > fabsf(sum))
			{
				sum = outputs[sourceindex + index];
				counter = J;
			}
		}

		K += indexwidth;
	}

	K = I;
	for (J = 0; J < indexheight; J++)
	{
		index = indexmap[K];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			if (counter == J)
				source[sourceindex + index] = result[resultindex + I];
			else
				source[sourceindex + index] = 0;
		}

		K += indexwidth;
	}
}

long cudaNeuralArrayIndexMap2DPoolingReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* outputs, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;

	if ((result == 0) || (source == 0) || (outputs == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;

	if (indexmap->indexWidth != resultlength)
		return 0;

	if (resultcount != sourcecount)
		return 0;

	if (cudaShapeCompare(&source->arrayShape, &outputs->arrayShape))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultlength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMap2DPoolingReverse << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		outputs->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__device__ cudaNeuralUnit indexMap2DConvolutionValue(
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long  indexoffset,
	unsigned long  sourceindex,
	unsigned long  sourcelength,
	unsigned long  filterindex,
	unsigned long  filterlength)
{
	unsigned long  I, J;
	unsigned long  index;
	cudaNeuralUnit sum;

	sum = 0;
	J = indexoffset;
	for (I = 0; I < indexheight; I++)
	{
		index = indexmap[J];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index] * filter[filterindex + I];
		}

		J += indexwidth;
	}

	return sum;
}

__global__ void kernelArrayIndexMap2DConvolutionForward(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I;
	unsigned long  J = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  K = blockDim.y * blockIdx.y + threadIdx.y;
	cudaNeuralUnit sum;

	if (J >= resultcount)
		return;
	if (K >= resultlength)
		return;

	unsigned long   resultindex = J * resultlength;
	unsigned long   sourceindex = 0;
	unsigned long   filterindex = (J * sourcecount) * filterlength;

	sum = 0;
	for (I = 0; I < sourcecount; I++)
	{
		sum += indexMap2DConvolutionValue(
			source,
			filter,
			indexwidth,
			indexheight,
			indexmap,
			K,
			sourceindex,
			sourcelength,
			filterindex,
			filterlength);

		sourceindex += sourcelength;
		filterindex += filterlength;
	}

	if ((sum < -3) || (sum > 3))
	{
		sum = 0;
		filterindex = (J * sourcecount) * filterlength;
		for (I = 0; I < sourcecount; I++)
		{
			for (J = 0; J < filterlength; J++)
				filter[filterindex + J] *= 0.9;

			filterindex += filterlength;
		}
	}

	result[resultindex + K] = sum;
}

long cudaNeuralArrayIndexMap2DConvolutionForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != resultlength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(resultcount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultlength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMap2DConvolutionForward << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMap2DConvolutionReverse(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J;
	unsigned long  K = blockDim.y * blockIdx.y + threadIdx.y;
	cudaNeuralUnit sum;

	if (I >= sourcecount)
		return;
	if (K >= sourcelength)
		return;

	unsigned long   resultindex = 0;
	unsigned long   sourceindex = I * sourcelength;
	unsigned long   filterindex = I * filterlength;
	unsigned long   filterjump = sourcecount * filterlength;

	sum = 0;
	for (J = 0; J < resultcount; J++)
	{
		sum += indexMap2DConvolutionValue(
			result,
			filter,
			indexwidth,
			indexheight,
			indexmap,
			K,
			resultindex,
			resultlength,
			filterindex,
			filterlength);

		resultindex += resultlength;
		filterindex += filterjump;
	}

	source[sourceindex + K] = sum;
}

long cudaNeuralArrayIndexMap2DConvolutionReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != sourcelength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(sourcecount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(sourcelength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayIndexMap2DConvolutionReverse << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayIndexMap2DConvolutionGetDerivatives(
	cudaNeuralUnit* result,
	cudaNeuralUnit* source,
	cudaNeuralUnit* filter,
	unsigned long   indexwidth,
	unsigned long   indexheight,
	unsigned long* indexmap,
	unsigned long resultlength,
	unsigned long resultcount,
	unsigned long sourcelength,
	unsigned long sourcecount,
	unsigned long filterlength,
	unsigned long filtercount)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned long  K = blockDim.z * blockIdx.z + threadIdx.z;

	if (I >= sourcecount)
		return;
	if (J >= resultcount)
		return;
	if (K >= filterlength)
		return;

	unsigned long   sourceindex = I * sourcelength;
	unsigned long   resultindex = J * resultlength;
	unsigned long   filterindex = (J * sourcecount + I) * filterlength;

	unsigned long  X, Y;
	unsigned long  index;
	unsigned long  counter;
	cudaNeuralUnit sum;

	sum = 0;
	counter = 0;
	Y = K * indexwidth;
	for (X = 0; X < indexwidth; X++)
	{
		index = indexmap[Y + X];
		if ((index != CUDA_INVALID_INDEX) &&
			(index < sourcelength))
		{
			sum += source[sourceindex + index] * result[resultindex + X];
			counter++;
		}
	}

	if (counter > 1)
		sum = sum / counter;

	filter[filterindex + K] = sum;

}

long cudaNeuralArrayIndexMap2DConvolutionGetDerivatives(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* filter, cudaIndexMap* indexmap)
{
	unsigned long   resultlength;
	unsigned long   resultcount;
	unsigned long   sourcelength;
	unsigned long   sourcecount;
	unsigned long   filterlength;
	unsigned long   filtercount;

	if ((result == 0) || (source == 0) || (filter == 0) || (indexmap == 0))
		return 0;

	resultlength = (result->arrayShape.shapeWidth * result->arrayShape.shapeHeight);
	resultcount = result->arrayShape.shapeDepth;
	sourcelength = (source->arrayShape.shapeWidth * source->arrayShape.shapeHeight);
	sourcecount = source->arrayShape.shapeDepth;;
	filterlength = (filter->arrayShape.shapeWidth * filter->arrayShape.shapeHeight);
	filtercount = filter->arrayShape.shapeDepth;;

	if (filtercount != (resultcount * sourcecount))
		return 0;

	if ((indexmap->indexWidth != resultlength) ||
		(indexmap->indexHeight != filterlength))
		return 0;

	dim3 threadsPerBlock(8, 8, 8);
	dim3 blocksPerGrid(
		(sourcecount + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(resultcount + threadsPerBlock.y - 1) / threadsPerBlock.y,
		(filterlength + threadsPerBlock.z - 1) / threadsPerBlock.z);

	kernelArrayIndexMap2DConvolutionGetDerivatives << < blocksPerGrid, threadsPerBlock >> > (
		result->arrayData,
		source->arrayData,
		filter->arrayData,
		indexmap->indexWidth,
		indexmap->indexHeight,
		indexmap->indexData,
		resultlength,
		resultcount,
		sourcelength,
		sourcecount,
		filterlength,
		filtercount);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixForward(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I;
	unsigned long  J = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  K;
	cudaNeuralUnit sum;

	if (J >= resultlength)
		return;

	sum = 0;
	K = J * sourcelength;
	for (I = 0; I < sourcelength; I++)
	{
		sum += source[I] * weights[K + I];
	}

	result[J] = sum;
}

long cudaNeuralArrayMatrixForward(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayMatrixForward << < blocksPerGrid, threadsPerBlock >> > 
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixReverse(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J;
	unsigned long  K;
	cudaNeuralUnit sum;

	if (I >= sourcelength)
		return;

	sum = 0;
	K = I;
	for (J = 0; J < resultlength; J++)
	{
		sum += result[J] * weights[K];

		K += sourcelength;
	}

	source[I] = sum;
}

long cudaNeuralArrayMatrixReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((source->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayMatrixReverse << < blocksPerGrid, threadsPerBlock >> >
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayMatrixGetDerivatives(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* weights, unsigned long resultlength, unsigned long sourcelength)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long  J = blockDim.y * blockIdx.y + threadIdx.y;

	if (I >= sourcelength)
		return;
	if (J >= resultlength)
		return;

	weights[J * sourcelength + I] = result[J] * source[I];
}

long cudaNeuralArrayMatrixGetDerivatives(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* weights)
{
	if ((result == 0) || (source == 0) || (weights == 0))
		return 0;

	if (weights->arrayLength != (result->arrayLength * source->arrayLength))
		return 0;

	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(
		(source->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(result->arrayLength + threadsPerBlock.y - 1) / threadsPerBlock.y);

	kernelArrayMatrixGetDerivatives << < blocksPerGrid, threadsPerBlock >> >
		(result->arrayData, source->arrayData, weights->arrayData, result->arrayLength, source->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__device__ double cuda_reluFunction(double a)
{
	if (a > 0)
		return a;

	return a * 0.01;
}

__device__ double cuda_reluDerivative(double a)
{
	if (a < 0)
		return 0.01;

	return 1;
}

__device__ double cuda_sigmoidFunction(double a)
{
	return 1 / (1 + exp(-a));
}

__device__ double cuda_sigmoidDerivative(double a)
{
	return (cuda_sigmoidFunction(a) * (1 - cuda_sigmoidFunction(a)));
}

__device__ double cuda_tanhFunction(double a)
{
	return tanh(a);
}

__device__ double cuda_tanhDerivative(double a)
{
	return 1 - (tanh(a) * tanh(a));
}

__global__ void kernelArrayTransferForward(cudaNeuralUnit* result, cudaNeuralUnit* source, unsigned long length, unsigned long transfer)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	switch (transfer)
	{
	case CUDA_TRANSFER_RELU:
		result[I] = cuda_reluFunction(source[I]);
	case CUDA_TRANSFER_SIGMOID:
		result[I] = cuda_sigmoidFunction(source[I]);
	case CUDA_TRANSFER_TANH:
		result[I] = cuda_tanhFunction(source[I]);
	}
}

long cudaNeuralArrayTransferForward(cudaNeuralArray* result, cudaNeuralArray* source, unsigned long transfer)
{
	if ((result == 0) || (source == 0))
		return 0;

	if (result->arrayLength != source->arrayLength)
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayTransferForward << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, source->arrayData, result->arrayLength, transfer);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayTransferReverse(cudaNeuralUnit* result, cudaNeuralUnit* source, cudaNeuralUnit* outputs, unsigned long length, unsigned long transfer)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	switch (transfer)
	{
	case CUDA_TRANSFER_RELU:
		source[I] = result[I] * cuda_reluDerivative(outputs[I]);
	case CUDA_TRANSFER_SIGMOID:
		source[I] = result[I] * cuda_sigmoidDerivative(outputs[I]);
	case CUDA_TRANSFER_TANH:
		source[I] = result[I] * cuda_tanhDerivative(outputs[I]);
	}
}

long cudaNeuralArrayTransferReverse(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* outputs, unsigned long transfer)
{
	if ((result == 0) || (source == 0) || (outputs == 0))
		return 0;

	if ((result->arrayLength != source->arrayLength) ||
		(result->arrayLength != outputs->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayTransferReverse << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, source->arrayData, outputs->arrayData, result->arrayLength, transfer);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayTransferFocus(
	cudaNeuralUnit* result, 
	cudaNeuralUnit* source, 
	cudaNeuralUnit* outputs, 
	cudaNeuralUnit* target, 
	cudaNeuralUnit  lambda, 
	cudaNeuralUnit  range, 
	unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	cudaNeuralUnit   gap, grad;

	gap = (target[I] - outputs[I]);
	grad = (gap / range) * (gap / range);
	source[I] = result[I] + (grad * lambda) * gap;
}

long cudaNeuralArrayTransferFocus(cudaNeuralArray* result, cudaNeuralArray* source, cudaNeuralArray* outputs, cudaNeuralArray* target, cudaNeuralUnit lambda, cudaNeuralUnit range)
{
	if ((result == 0) || (source == 0) || (outputs == 0) || (target == 0))
		return 0;

	if ((result->arrayLength != source->arrayLength) ||
		(result->arrayLength != outputs->arrayLength) ||
		(result->arrayLength != target->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayTransferFocus << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, source->arrayData, outputs->arrayData, target->arrayData, lambda, range, result->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayUpdateMomentum(cudaNeuralUnit* weights, cudaNeuralUnit* vectors, cudaNeuralUnit* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

	if (I >= length)
		return;

	vectors[I] = (vectors[I] * momentum) + (1 - momentum) * (deltas[I]);

	weights[I] = weights[I] + learningrate * vectors[I];
}

long cudaNeuralArrayUpdateMomentum(cudaNeuralArray* weights, cudaNeuralArray* vectors, cudaNeuralArray* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum)
{
	if ((weights == 0) || (vectors == 0) || (deltas == 0))
		return 0;

	if ((weights->arrayLength != vectors->arrayLength) ||
		(weights->arrayLength != deltas->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((weights->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayUpdateMomentum << < blocksPerGrid, threadsPerBlock >> > 
		(weights->arrayData, vectors->arrayData, deltas->arrayData, learningrate, momentum, weights->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayUpdateAdagrad(cudaNeuralUnit* weights, cudaNeuralUnit* vectors, cudaNeuralUnit* gammas, cudaNeuralUnit* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	cudaNeuralUnit epsilon = 0.00000001;
	cudaNeuralUnit m2 = 0.999;
	cudaNeuralUnit grad;

	if (I >= length)
		return;

	gammas[I] = (gammas[I] * m2) + (1 - m2) * (deltas[I] * deltas[I]);
	grad = (learningrate / (sqrt(gammas[I]) + epsilon));

	vectors[I] = (vectors[I] * momentum) + (1 - momentum) * (deltas[I]);

	weights[I] = weights[I] + grad * vectors[I];
}

long cudaNeuralArrayUpdateAdagrad(cudaNeuralArray* weights, cudaNeuralArray* vectors, cudaNeuralArray* gammas, cudaNeuralArray* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum)
{
	if ((weights == 0) || (vectors == 0) || (deltas == 0))
		return 0;

	if ((weights->arrayLength != vectors->arrayLength) ||
		(weights->arrayLength != gammas->arrayLength) || 
		(weights->arrayLength != deltas->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((weights->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayUpdateAdagrad << < blocksPerGrid, threadsPerBlock >> >
		(weights->arrayData, vectors->arrayData, gammas->arrayData, deltas->arrayData, learningrate, momentum, weights->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelArrayUpdateAdam(cudaNeuralUnit* weights, cudaNeuralUnit* vectors, cudaNeuralUnit* gammas, cudaNeuralUnit* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum, unsigned long length)
{
	unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;
	cudaNeuralUnit epsilon = 0.00000001;
	cudaNeuralUnit m2 = 0.999;
	cudaNeuralUnit vt, mt;
	cudaNeuralUnit grad;

	if (I >= length)
		return;

	gammas[I] = (gammas[I] * m2) + (1 - m2) * (deltas[I] * deltas[I]);
	vt = gammas[I] / (1 - m2);

	vectors[I] = (vectors[I] * momentum) + (1 - momentum) * (deltas[I]);
	mt = vectors[I] / (1 - momentum);

	grad = (learningrate / (sqrt(vt) + epsilon));

	weights[I] = weights[I] + (mt * grad);
}

long cudaNeuralArrayUpdateAdam(cudaNeuralArray* weights, cudaNeuralArray* vectors, cudaNeuralArray* gammas, cudaNeuralArray* deltas, cudaNeuralUnit learningrate, cudaNeuralUnit momentum)
{
	if ((weights == 0) || (vectors == 0) || (deltas == 0))
		return 0;

	if ((weights->arrayLength != vectors->arrayLength) ||
		(weights->arrayLength != gammas->arrayLength) ||
		(weights->arrayLength != deltas->arrayLength))
		return 0;

	dim3 threadsPerBlock(512);
	dim3 blocksPerGrid((weights->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

	kernelArrayUpdateAdam << < blocksPerGrid, threadsPerBlock >> >
		(weights->arrayData, vectors->arrayData, gammas->arrayData, deltas->arrayData, learningrate, momentum, weights->arrayLength);

	return cudaMemoryDeviceSynchronize();
}

long cudaNeuralArrayDestroy(cudaNeuralArray* parent)
{
	if (parent == 0)
		return 0;

	cudaMemoryFreeMemory((void*)parent->arrayData);

	parent->arrayLength = 0;
	parent->arraySize = 0;
	parent->arrayData = 0;

	return 1;
}

#endif // CUDANEURALARRAY_H

