#include "hip/hip_runtime.h"
﻿#define CUDA_LIBRARY_KERNEL

#include "cudalibrary.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#ifndef CUDAMEMORY_H
#define CUDAMEMORY_H

long cudaMemoryResetDevice()
{
    hipError_t cudaStatus;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceReset failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemorySetDevice(unsigned long deviceID)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(deviceID);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemoryAllocateMemory(void** result, unsigned long length)
{
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)result, length);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cudaMemoryAllocateMemory failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemorySetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
    hipError_t cudaStatus;

    cudaStatus = hipMemcpy(devicememory, hostmemory, length, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cudaMemorySetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemoryGetDeviceMemory(void* devicememory, void* hostmemory, unsigned long length)
{
    hipError_t cudaStatus;

    cudaStatus = hipMemcpy(hostmemory, devicememory, length, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cudaMemoryGetDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemoryCopyDeviceMemory(void* devicedestination, void* devicesource, unsigned long length)
{
    hipError_t cudaStatus;

    cudaStatus = hipMemcpy(devicedestination, devicesource, length, hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cudaMemoryCopyDeviceMemory failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemoryDeviceSynchronize()
{
    hipError_t cudaStatus;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cudaMemoryDeviceSynchronize failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

long cudaMemoryFreeMemory(void* source)
{
    hipError_t cudaStatus;

    cudaStatus = hipFree(source);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemoryFreeMemory failed! %s\n", hipGetErrorString(cudaStatus));

        return 0;
    }

    return 1;
}

#endif // CUDAMEMORY_H

#ifndef CUDANEURALARRAY_H
#define CUDANEURALARRAY_H

#include "cudalibrary.h"

#include <stdlib.h>

long cudaNeuralArrayCreate(cudaNeuralArray* parent, unsigned long width, unsigned long height, unsigned long depth)
{
    cudaNeuralUnit* data;
    unsigned long   length;
    unsigned long   datalength;

    if (parent == 0)
        return 0;

    typeShapeAssign(&parent->arrayShape, width, height, depth);
    length = typeShapeGetLength(&parent->arrayShape);
    datalength = sizeof(cudaNeuralUnit) * length;

    if (cudaMemoryAllocateMemory((void**)&data, datalength))
    {
        parent->arrayLength = length;
        parent->arraySize = datalength;
        parent->arrayData = data;

        return 1;
    }

    typeShapeZero(&parent->arrayShape);
    parent->arrayLength = 0;
    parent->arraySize = 0;
    parent->arrayData = 0;

    return 0;
}

long cudaNeuralArraySetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
    if ((parent == 0) || (buffer == 0))
        return 0;

    if (buffersize != parent->arrayLength)
        return 0;

    return cudaMemorySetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayGetData(cudaNeuralArray* parent, cudaNeuralUnit* buffer, unsigned long buffersize)
{
    if ((parent == 0) || (buffer == 0))
        return 0;

    if (buffersize != parent->arrayLength)
        return 0;

    return cudaMemoryGetDeviceMemory(parent->arrayData, buffer, buffersize * sizeof(cudaNeuralUnit));
}

long cudaNeuralArrayCopy(cudaNeuralArray* parent, cudaNeuralArray* source)
{
    if ((parent == 0) || (source == 0))
        return 0;

    if (parent->arrayLength != source->arrayLength)
        return 0;

    return cudaMemoryCopyDeviceMemory(parent->arrayData, source->arrayData, parent->arrayLength * sizeof(cudaNeuralUnit));
}

long cudaNeuralArraySetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
    unsigned long   I, J;
    unsigned long   A, B, C;
    unsigned long   length;
    unsigned char*  color;
    cudaNeuralUnit* buffer;

    if ((parent == 0) || (imagedata == 0))
        return 0;

    if (typeShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
        return 0;

    color = (unsigned char*)imagedata;
    buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
    if (buffer == 0)
        return 0;

    length = imagewidth * imageheight;
    A = length * 0;
    B = length * 1;
    C = length * 2;

    J = 0;
    for (I = 0; I < length; I++)
    {
        buffer[A + I] = (cudaNeuralUnit)color[J + 0] / 255;
        buffer[B + I] = (cudaNeuralUnit)color[J + 1] / 255;
        buffer[C + I] = (cudaNeuralUnit)color[J + 2] / 255;

        J += 4;
    }

    cudaNeuralArraySetData(parent, buffer, parent->arrayLength);

    free(buffer);

    return 1;
}

double neural_maxout(double a)
{
    if (a < 0)
        return 0;

    if (a > 1)
        return 1;

    return a;
}

long cudaNeuralArrayGetImageRGB(cudaNeuralArray* parent, unsigned long imagewidth, unsigned long imageheight, unsigned long* imagedata)
{
    unsigned long   I, J;
    unsigned long   A, B, C;
    unsigned long   length;
    unsigned char* color;
    cudaNeuralUnit* buffer;

    if ((parent == 0) || (imagedata == 0))
        return 0;

    if (typeShapeCompareValue(&parent->arrayShape, imagewidth, imageheight, 3) == 0)
        return 0;

    color = (unsigned char*)imagedata;
    buffer = (cudaNeuralUnit*)malloc(parent->arraySize);
    if (buffer == 0)
        return 0;

    cudaNeuralArraySetData(parent, buffer, parent->arrayLength);

    length = imagewidth * imageheight;
    A = length * 0;
    B = length * 1;
    C = length * 2;

    J = 0;
    for (I = 0; I < length; I++)
    {
        color[J + 0] = (unsigned char)(neural_maxout(buffer[A + I]) * 255);
        color[J + 1] = (unsigned char)(neural_maxout(buffer[B + I]) * 255);
        color[J + 2] = (unsigned char)(neural_maxout(buffer[C + I]) * 255);

        J += 4;
    }

    free(buffer);

    return 1;
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillZero(cudaNeuralUnit* result, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = 0;
}

long cudaNeuralArrayFillZero(cudaNeuralArray* result)
{
    if (result == 0)
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelFillZero << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelFillOnes(cudaNeuralUnit* result, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = 1;
}

long cudaNeuralArrayFillOnes(cudaNeuralArray* result)
{
    if (result == 0)
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelFillOnes << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void kernelFillRandom(cudaNeuralUnit* result, unsigned long seed, unsigned long length)
{
    hiprandState    state;
    unsigned long  I;

    hiprand_init(seed, 0, 0, &state);

    for (I = 0; I < length; I++)
    {
        result[I] = hiprand_uniform(&state);
    }
}

long cudaNeuralArrayFillRandom(cudaNeuralArray* result)
{
    if (result == 0)
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelFillRandom << < 1, 1 >> > (result->arrayData, clock(), result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelAdd(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = value1[I] + value2[I];
}

long cudaNeuralArrayAdd(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
    if ((result == 0) || (value1 == 0) || (value2 == 0))
        return 0;

    if ((result->arrayLength != value1->arrayLength) ||
        (result->arrayLength != value2->arrayLength))
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelAdd << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelSubtract(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = value1[I] - value2[I];
}

long cudaNeuralArraySubtract(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
    if ((result == 0) || (value1 == 0) || (value2 == 0))
        return 0;

    if ((result->arrayLength != value1->arrayLength) ||
        (result->arrayLength != value2->arrayLength))
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelSubtract << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelMultiply(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = value1[I] * value2[I];
}

long cudaNeuralArrayMultiply(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
    if ((result == 0) || (value1 == 0) || (value2 == 0))
        return 0;

    if ((result->arrayLength != value1->arrayLength) ||
        (result->arrayLength != value2->arrayLength))
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelMultiply << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelDivide(cudaNeuralUnit* result, cudaNeuralUnit* value1, cudaNeuralUnit* value2, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    if (value2[I] != 0)
        result[I] = value1[I] * value2[I];
    else
        result[I] = 0;
}

long cudaNeuralArrayDivide(cudaNeuralArray* result, cudaNeuralArray* value1, cudaNeuralArray* value2)
{
    if ((result == 0) || (value1 == 0) || (value2 == 0))
        return 0;

    if ((result->arrayLength != value1->arrayLength) ||
        (result->arrayLength != value2->arrayLength))
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelDivide << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, value1->arrayData, value2->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

//////////////////////////////////////////////////////////////////////////////////
__global__ void kernelReverse(cudaNeuralUnit* result, cudaNeuralUnit* source, unsigned long length)
{
    unsigned long  I = blockDim.x * blockIdx.x + threadIdx.x;

    if (I >= length)
        return;

    result[I] = source[length - I - 1];
}

long cudaNeuralArrayReverse(cudaNeuralArray* result, cudaNeuralArray* source)
{
    if ((result == 0) || (source == 0))
        return 0;

    if (result->arrayLength != source->arrayLength)
        return 0;

    dim3 threadsPerBlock(512);
    dim3 blocksPerGrid((result->arrayLength + threadsPerBlock.x - 1) / threadsPerBlock.x);

    kernelReverse << < blocksPerGrid, threadsPerBlock >> > (result->arrayData, source->arrayData, result->arrayLength);

    return cudaMemoryDeviceSynchronize();
}

long cudaNeuralArrayDestroy(cudaNeuralArray* parent)
{
    if (parent == 0)
        return 0;

    cudaMemoryFreeMemory((void*)parent->arrayData);

    parent->arrayLength = 0;
    parent->arraySize = 0;
    parent->arrayData = 0;

    return 1;
}

#endif // CUDANEURALARRAY_H

